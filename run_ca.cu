#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <stdlib.h>
#include <iterator> 

#include <png++/png.hpp>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "ca.cu"

// Random number in range functor
struct randomInts{

    private:
        const int a;
    
    public:
        randomInts(int A):a(A){}
    
        __host__ __device__
        int operator()(){ return (int)std::floor((float)(a)*rand()/RAND_MAX); }
};

// Print thrust vector values to console
template<class T>
void consolePrinter(T& x, std::string m, const char* seperator)
{
    std::cout << m;
    thrust::copy(x.begin(),x.end(),std::ostream_iterator<int>(std::cout,seperator));
    std::cout << "\n";
}

// Print values to grayscale png
template <class T,class I>
void pngPrinter(T& x,int n, I& image)
{   
    thrust::copy(x.begin(),x.end(),image[n].begin());
}

int main(int argc, char* argv[])
{    
    int range  = 3;             // Cell neighbour view range (default for now)
    int states = atoi(argv[3]); // Number of states
    int length = atoi(argv[1]); // Length of cell array
    int steps  = atoi(argv[2]); // Number of update steps
    
    // Rule number: The possible values for this turns out 
    // to grow very quickly considering for example for
    // three cells only this is already n^(n^3)
    unsigned long int rule = std::stol(argv[4]); 
    
    int statePerms = pow(states,range); // Number of possible permutations of cells

    if ( rule >=  pow(states,statePerms))
    {
        std::cout << "Rule outside range\n";
        return 10; 
    }

    // PNG image storage    
    png::image< png::gray_pixel > image(length,steps);

    thrust::host_vector<int> init(length);
    
    // Generate random initial cell states on host
    thrust::generate(init.begin(),init.end(),randomInts(states));
    
    /* BELOW CAN SET SPECIFIC CELLS (SET AS ARGUMENT)
    thrust::fill(init.begin(),init.end(),0);
    int mid = length/2;
    init[mid] = 1;
    init[mid+2] = 1;
    */

    // Populate ruleset vector on host
    thrust::host_vector<int> rules(statePerms);
    unsigned long int x = rule;
    for(int i=0; i<statePerms; ++i)
    {
        rules[i] = x % states;
        x = x / (unsigned long int)states;
    }
    consolePrinter(rules,"Ruleset:|","|");
    
    // Load ruleset into device vector
    thrust::device_vector<int> d_rules(statePerms);
    thrust::copy(rules.begin(),rules.end(),d_rules.begin());

    // Colour transformation from states number to 8-bit grayscale
    thrust::device_vector<int> clr(length);
    thrust::fill(clr.begin(),clr.end(),255/(states-1));

    // Initialize functor
    caUpdate CA(thrust::raw_pointer_cast(&d_rules[0]),states);
    
    // Time step counter
    int counter = 0;

    // Initialize CA object
    ca1d caa(length,states,CA);
    caa.loadInitial(init);

    while(counter < steps){

        // Update front array from back
        caa.updateFront();
        
        // Update colour value vector and copy to PNG
        thrust::transform(caa.ft.begin(),caa.ft.end(),clr.begin(),outVec.begin(),thrust::multiplies<short int>());
        pngPrinter(outVec,counter,image);

        // Swap front and back, increment
        caa.swapFB();
        ++counter;
    }
    
    // Save PNG data
    image.write("pngs/rule_"+std::string(argv[4])+"_"+std::string(argv[3])+".png");

    return 0;
}