#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/for_each.h>

// CA Update process functor
struct caUpdate{
    
    private:
        const int* ruleArr;
        int states;

    public:
        caUpdate(){}
        caUpdate(int* x, int s):ruleArr(x),states(s){}
        
        template <class Tuple>
        __device__
        void operator()(Tuple t)
        {
            int a = thrust::get<0>(t);  // Left cell 
            int b = thrust::get<1>(t);  // This cell
            int c = thrust::get<2>(t);  // Right cell
            thrust::get<3>(t) = *(ruleArr+b+states*a+states*states*c);
        }
};

class ca1d{

    typedef thrust::device_vector<int> intDvec; 

        int length;      // Array length of CA
        int states;      // CA number of states
        intDvec bk;      // Back (past) array
        intDvec rt;      // Right neighbour map
        intDvec lt;      // Left neighbour map
        caUpdate update; // Update functor
    
    public:
        intDvec ft;      // Front (current) array

        ca1d(int l, int s, caUpdate ca):length(l),states(s)
        {
            bk.resize(length);
            ft.resize(length);
            lt.resize(length);
            rt.resize(length);
            update = ca;
            // Load maps to neighbouring cells (wrapping ends)
            thrust::counting_iterator<int> it(0);
            lt[0] = length-1;
            thrust::copy(it,it+length,lt.begin()+1);
            thrust::copy(it+1,it+length,rt.begin());
            rt[length-1] = 0;
        }

        void loadInitial(thrust::host_vector<int> i)
        {
            thrust::copy(i.begin(),i.end(),bk.begin());
        }

        void updateFront()
        {
            thrust::for_each(
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                        bk.begin(),
                        thrust::make_permutation_iterator(bk.begin(),lt.begin()),
                        thrust::make_permutation_iterator(bk.begin(),rt.begin()),
                        ft.begin()
                    )
                ),
                thrust::make_zip_iterator(
                    thrust::make_tuple(
                        bk.end(),
                        thrust::make_permutation_iterator(bk.begin(),lt.end()),
                        thrust::make_permutation_iterator(bk.begin(),rt.end()),
                        ft.end()
                    )
                ),
                update
            );
        }

        void swapFB()
        {
            ft.swap(bk);
        }
};